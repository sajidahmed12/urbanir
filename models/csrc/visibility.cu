#include "hip/hip_runtime.h"
#include "utils.h"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

template <typename scalar_t>
__global__ void composite_weight_test_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> alive_indices,
    const scalar_t T_threshold,
    const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> N_eff_samples,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> opacity,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> ws
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= alive_indices.size(0)) return;
    if (N_eff_samples[n]==0) return; // no hit

    const size_t r = alive_indices[n]; // ray index

    // front to back compositing
    int s = 0; scalar_t T = 1-opacity[r];

    while (s < N_eff_samples[n]) {
        const scalar_t a = 1.0f - __expf(-sigmas[n][s]*deltas[n][s]);
        const scalar_t w = a * T;
        ws[n][s] = w;
        T *= 1.0f-a;
        if (T <= T_threshold) break; // ray has enough opacity
        s++;
    }
}

torch::Tensor composite_weight_test_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor deltas,
    const torch::Tensor alive_indices,
    const float T_threshold,
    const torch::Tensor N_eff_samples,
    const torch::Tensor opacity
){
    const int N = sigmas.size(0);
    const int S = sigmas.size(1);
    auto ws = torch::zeros({N, S}, sigmas.options());

    const int N_rays = alive_indices.size(0);
    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_weight_test_fw_cu", 
    ([&] {
        composite_weight_test_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            alive_indices.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
            T_threshold,
            N_eff_samples.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
            opacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return ws;
}

template <typename scalar_t>
__global__ void composite_visibility_test_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> viss_t,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> deltas,
    torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> alive_indices,
    const scalar_t T_threshold,
    const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> N_eff_samples,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> opacity,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> visibility_T
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= alive_indices.size(0)) return;

    if (N_eff_samples[n]==0){ // no hit
        alive_indices[n] = -1;
        return;
    }

    const size_t r = alive_indices[n]; // ray index

    // front to back compositing
    int s = 0; scalar_t T = 1-opacity[r];

    while (s < N_eff_samples[n]) {
        const scalar_t a = 1.0f - __expf(-sigmas[n][s]*deltas[n][s]);
        const scalar_t w = a * T;
        visibility_T[r] += w*viss_t[n][s];
        T *= 1.0f-a;
        if (T <= T_threshold){ // ray has enough opacity
            alive_indices[n] = -1;
            break;
        }
        s++;
    }
}

void composite_visibility_test_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor viss_t,
    const torch::Tensor deltas,
    torch::Tensor alive_indices,
    const float T_threshold,
    const torch::Tensor N_eff_samples,
    torch::Tensor opacity,
    torch::Tensor visibility_T
){
    const int N_rays = alive_indices.size(0);

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "composite_visibility_test_fw_cu", 
    ([&] {
        composite_visibility_test_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            viss_t.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            alive_indices.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
            T_threshold,
            N_eff_samples.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
            opacity.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            visibility_T.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));
}


template <typename scalar_t>
__global__ void visibility_train_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> light_dist,
    const scalar_t T_threshold,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> visibility,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // front to back compositing
    int samples = 0; scalar_t T = 1.0f;

    while (samples < N_samples) {
        const int s = start_idx + samples;
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        const scalar_t w = a * T;

        if (ts[s] > light_dist[ray_idx]) break;

        visibility[ray_idx] -= w;
        ws[s] = w;
        T *= 1.0f-a;

        if (T <= T_threshold) break; // ray has enough opacity
        samples++;
    }
}


std::vector<torch::Tensor> visibility_train_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const torch::Tensor light_dist,
    const float T_threshold
){
    const int N = sigmas.size(0), N_rays = rays_a.size(0);
    auto visibility = torch::ones({N_rays}, sigmas.options());
    auto ws = torch::zeros({N}, sigmas.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "visibility_train_fw_cu", 
    ([&] {
        visibility_train_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            light_dist.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            T_threshold,
            visibility.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {visibility, ws};
}

template <typename scalar_t>
__global__ void visibility_train_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dvisibility,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dws,
    scalar_t* __restrict__ dL_dws_times_ws,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> visibility,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> light_dist,
    const scalar_t T_threshold,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dsigmas
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // front to back compositing
    int samples = 0;
    scalar_t V = visibility[ray_idx];
    scalar_t T = 1.0f;

    // compute prefix sum of dL_dws * ws
    // [a0, a1, a2, a3, ...] -> [a0, a0+a1, a0+a1+a2, a0+a1+a2+a3, ...]
    thrust::inclusive_scan(thrust::device,
                           dL_dws_times_ws+start_idx,
                           dL_dws_times_ws+start_idx+N_samples,
                           dL_dws_times_ws+start_idx);
    scalar_t dL_dws_times_ws_sum = dL_dws_times_ws[start_idx+N_samples-1];

    while (samples < N_samples) {
        const int s = start_idx + samples;
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        const scalar_t w = a * T;

        if (ts[s] > light_dist[ray_idx]) break;

        T *= 1.0f-a;

        dL_dsigmas[s] = deltas[s] * (
            dL_dvisibility[ray_idx]*(-V) +
            T*dL_dws[s]-(dL_dws_times_ws_sum-dL_dws_times_ws[s])
        );

        if (T <= T_threshold) break; // ray has enough opacity
        samples++;
    }
}


torch::Tensor visibility_train_bw_cu(
    const torch::Tensor dL_dvisibility,
    const torch::Tensor dL_dws,
    const torch::Tensor sigmas,
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const torch::Tensor visibility,
    const torch::Tensor light_dist,
    const float T_threshold
){
    const int N = sigmas.size(0), N_rays = rays_a.size(0);

    auto dL_dsigmas = torch::zeros({N}, sigmas.options());
    auto dL_dws_times_ws = dL_dws * ws; // auxiliary input

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "visibility_train_bw_cu", 
    ([&] {
        visibility_train_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dvisibility.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_dws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            dL_dws_times_ws.data_ptr<scalar_t>(),
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            visibility.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            light_dist.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            T_threshold,
            dL_dsigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return dL_dsigmas;
}

template <typename scalar_t>
__global__ void visibility_test_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> ts,
    torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> alive_indices,
    const scalar_t T_threshold,
    const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> N_eff_samples,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> light_dist,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> visibility
    
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= alive_indices.size(0)) return;

    if (N_eff_samples[n]==0){ // no hit
        alive_indices[n] = -1;
        return;
    }

    const size_t r = alive_indices[n]; // ray index

    // front to back compositing
    int s = 0; scalar_t T = visibility[r];

    while (s < N_eff_samples[n]) {
        if (ts[n][s] > light_dist[r]){ // ray reach light source
            alive_indices[n] = -1;
            break;
        }

        const scalar_t a = 1.0f - __expf(-sigmas[n][s]*deltas[n][s]);
        const scalar_t w = a * T;

        visibility[r] -= w;
        
        T *= 1.0f-a;
        if (T <= T_threshold){ // ray has enough opacity
            alive_indices[n] = -1;
            break;
        }        
        s++;
    }
}

void visibility_test_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    torch::Tensor alive_indices,
    const float T_threshold,
    const torch::Tensor N_eff_samples,
    const torch::Tensor light_dist,
    torch::Tensor visibility
){
    const int N_rays = alive_indices.size(0);

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "visibility_test_fw_cu", 
    ([&] {
        visibility_test_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            alive_indices.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
            T_threshold,
            N_eff_samples.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
            light_dist.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            visibility.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));
}
